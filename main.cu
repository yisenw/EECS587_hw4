
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <string.h>

using namespace std;
static const long long n = 1000;
int t;


__global__ void MatrixUpdate(double* A, double* B)
{
	long long idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n*n) {
		long long i = idx / n;
		long long j = idx % n;
		if (i == 0 || i == n-1 || j == 0 || j == n-1) B[idx] = A[idx];
		else 
		{
			auto t1 = max(min(A[idx-1],A[idx+1]),min(A[idx-n],A[idx+n]));
			auto t2 = min(max(A[idx-1],A[idx+1]),max(A[idx-n],A[idx+n]));
			B[idx] = max(min(t1,t2),min(A[idx],max(t1,t2)));
		}
	}
}

__global__ void MatrixVerify1(double* A, double* C)
{
	long long idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx == 0) {
		C[2] = A[19*n+37];
		C[1] = A[(n/3)*n+(n/3)];
	}
}

__global__ void MatrixSum(double* A, double* C, long long * D)
{
	long long idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < n*n) {
		if (idx % (2 * D[0]) == 0 && idx + D[0] < n*n) {
			A[idx] += A[idx+D[0]];
		}
	}
}

__global__ void MatrixVerify2(double* A, double* C)
{
	long long idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx == 0) {
		C[0] = A[0];
	}
}



int main(int argc, char *argv[])
{
    t = atoi(argv[1]);    
	double* d_A;
	double* d_B;
    double* d_final;
	double* d_C;
	long long* d_D;
	auto size = n * n * sizeof(double);
	hipMalloc(&d_A, size);
	hipMalloc(&d_B, size);
	hipMalloc(&d_C, 3 * sizeof(double));
	hipMalloc(&d_D, sizeof(long long));
	double h_A[n * n];
	double h_C[3];
	for (long long k = 0; k < n * n; k++)
	{
		double i = (double) (k / n);
		double j = (double) (k % n);
		h_A[k] = sin(i*i+j) * sin(i*i+j) + cos(i-j);
	}
	
	hipEvent_t start, stop;
	float ttime;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipEventSynchronize(start);
	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	for (int i = 0; i < t/2; ++i){
		MatrixUpdate<<< (n*n+255)/256, 256 >>>(d_A, d_B);
		hipDeviceSynchronize();
		MatrixUpdate<<< (n*n+255)/256, 256 >>>(d_B, d_A);
		hipDeviceSynchronize();
	}

    if (t % 2 == 1) {
        MatrixUpdate<<< (n*n+255)/256, 256 >>>(d_A, d_B);
		hipDeviceSynchronize();
        d_final = d_B;
    }
    else d_final = d_A;

	MatrixVerify1<<< (n*n+255)/256, 256 >>>(d_final, d_C);
	hipDeviceSynchronize();
	long long st = 1;
	while (st <= n*n)
	{
		hipMemcpy(d_D, &st, sizeof(long long), hipMemcpyHostToDevice);
		MatrixSum<<< (n*n+255)/256, 256 >>>(d_final, d_C, d_D);
		hipDeviceSynchronize();
		st *= 2;
	}
	MatrixVerify2<<< (n*n+255)/256, 256 >>>(d_final, d_C);
	hipDeviceSynchronize();
	hipMemcpy(h_C, d_C, 3 * sizeof(double), hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&ttime, start, stop);
	cout << "sum: " << (long long) h_C[0] << "\n";
	cout << "A[n/3, n/3] = " << h_C[1] << "\n";
	cout << "A[19, 37] = " << h_C[2] << "\n";
	cout << "Time: " << ttime << " milliseconds";
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
	return 0;
}
